#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>

#define iterations 256
#define results_size iterations * iterations * iterations
#define frequencies_group_by_size iterations * iterations * iterations * 2


struct Offset {
    int x;
    int y;
    int z;
};

__global__ void add_kernel(unsigned int *results,  int x_offset, int y_offset, int z_offset) {
    int i = blockIdx.x;
    int j = threadIdx.x;


    // Largest number that i and j can be is 255, the offset allows the number to be larger
    // do to limits in number of threads cuda allows
    unsigned int x = i + (x_offset * iterations);
    unsigned int y = j + (y_offset * iterations);

    unsigned int result;

    // thread_array is an independent part of the results array for each thread that each thread can use,
    // to prevent race conditions.
    
    // All CUDA cores uses the same 1D array, but each core needs to have it own index-range in the array,
    // the core is the only one that can modify that part of the array.
    // Each result in the array is followed by its frequncy
    // Each thread in the array can have max 256 results
    // Each block has 256 threads
    // So to get block number 2 you need start pointer +=  (2 * iterations * iterations * 1)
    // where 1 is the index of block 2.
    unsigned int *thread_array = &results[((2 * iterations * j) + (2 * iterations * iterations * i))];

    int next_free_index = 0;
    int z = 0;

    for (int k=0; k < iterations; k++) {
        z = k + (z_offset * iterations);
        
        // The function the is being computed
        result = (x & y) ^ ((!x) & z);


        for (int l=0; l <= next_free_index; l++) {
            if (l == next_free_index) {
                // First instance of the result 

                next_free_index ++;

                // Assign the result to free location
                thread_array[(l * 2)] = result;

                // Initialize the counter for that particular result
                thread_array[(l * 2 + 1)] = 1;
                break;
            } else if(thread_array[(l * 2)] == result){
                // The result exits

                // Iterate the counter for that particular result
                thread_array[(l * 2 + 1)] ++;
                break;
            }
        }
    }
}

__global__ void merge_threads_arrays(unsigned int *frequencies_group_by_threads, unsigned int* frequencies_group_by_block) {
    // This function merges the result for each thread in the same block

    // Input:
    //  [[BLOCK 1], [BLOCK 2], ...]
    //  BLOCK 1 -> [[Thread 1], [Thread 2], ...]
    //  Thread 1 -> [[result 1, count], [result 2, count]]
    // The array is 1-dimensional but it is better to think about it as a 3-dimensional array

    // Output:
    //  [[BLOCK 1], [BLOCK 2], ...]
    //  BLOCK 1 -> [[result 1, count], [result 2, count], ...]
    // The array is 1-dimensional but it is better to think about it as a 3-dimensional array

    int i = blockIdx.x;

    unsigned int result, result_frequency;

    // Keeps track of the current thread in block i
    unsigned int* current_thread = frequencies_group_by_threads + (i * iterations * iterations * 2);

    // Start index of the array that the threads in block i are merge into
    unsigned int* block_frequencies = frequencies_group_by_block + (i * iterations * iterations * 2);

    int next_free_index = 0;

    for (int thread_index=0; thread_index < iterations; thread_index++) {
        for (int result_index=0; result_index < iterations; result_index++) {

            result = current_thread[result_index * 2];
            result_frequency = current_thread[result_index * 2 + 1];


            if(result_frequency == 0) break;

            for (int ii=0; ii <= next_free_index; ii++) {

                if (ii == next_free_index) {
                    // First instance of the result

                    next_free_index ++;
                    block_frequencies[ii * 2] = result;
                    block_frequencies[ii * 2 + 1] = result_frequency;
                    break;
                } else if(block_frequencies[ii * 2] == result) {

                    // The result exits
                    
                    block_frequencies[ii * 2 + 1] += result_frequency;
                    break;
                }
            }
        }

        // Each thread has 256 results, each result has 2 values (result, count)
        // so each thread has total 256 * 2 entries.
        current_thread +=  iterations * 2;
    }
}


int start_brute_force (int *results, unsigned int *frequencies, int inner_iterations) {
    unsigned int *cuda_frequencies_count = nullptr;
    unsigned int *cuda_block_frequencies_count = nullptr;

    unsigned int *frequencies_count = (unsigned int *) malloc(frequencies_group_by_size * sizeof(int));
    unsigned int *frequencies_group_by_blocks = (unsigned int*) malloc(frequencies_group_by_size* sizeof(int));

    hipMalloc((void **) &cuda_frequencies_count, frequencies_group_by_size * sizeof(int));
    hipMalloc((void **) &cuda_block_frequencies_count, frequencies_group_by_size * sizeof(int));

    Offset offset;

    // These 3 for loops create an offset so the program is able to run numbers > 256
    for (int i=0; i < inner_iterations; i++) {
        for (int j=0; j < inner_iterations; j++) {
            for (int k=0; k < inner_iterations; k++) {
                offset = {i, j, k};

                // Clear memory before using it again.
                hipMemset(cuda_frequencies_count, 0, frequencies_group_by_size * sizeof(int));
                hipMemset(cuda_block_frequencies_count, 0, frequencies_group_by_size * sizeof(int));
                memset(frequencies_group_by_blocks, 0, frequencies_group_by_size * sizeof(int));

                add_kernel<<<iterations, iterations>>>(cuda_frequencies_count, offset.x, offset.y, offset.z);

                hipDeviceSynchronize();

                hipMemcpy(frequencies_count, cuda_frequencies_count, frequencies_group_by_size * sizeof(int), hipMemcpyDeviceToHost);

                // There frequencies calulate are group by blockId and threadId, and need to be merge together
                // merge_threads_arrays merges the frequencies of all threads within a block to a single result
                merge_threads_arrays<<<256, 1>>>(cuda_frequencies_count, cuda_block_frequencies_count);

                hipDeviceSynchronize();

                hipMemcpy(frequencies_group_by_blocks, cuda_block_frequencies_count, frequencies_group_by_size * sizeof(int), hipMemcpyDeviceToHost);
                unsigned int result, result_frequency;

                unsigned int* current_block;

                // Merge each block array to a single result array
                for (int block_index=0; block_index < iterations; block_index++) {

                    current_block = frequencies_group_by_blocks + (block_index * iterations * iterations * 2);
                    for (int i=0; i < iterations * iterations; i++){
                        result = current_block[i * 2];
                        result_frequency = current_block[i * 2 + 1];

                        // If frequency is 0 there are no more result left in the current block
                        if(result_frequency == 0) break;
                        
                        // Sum all results by index as the result and value as frequency
                        frequencies[result] += result_frequency;
                    }
                }

            }
        }
    }
    hipFree(cuda_block_frequencies_count);
    hipFree(cuda_frequencies_count);
    free(frequencies_count);

    return 0;
}


int main(int argc, char * argv[]) {
    int N = atoi(argv[1]);

    unsigned int start_number = 0;
    unsigned int end_number = N;
    unsigned int range_size = end_number - start_number;

    // For timing the program
    clock_t start, end;
    double run_time;
    
    
    // Each CUDA iterations can run maxmium of N=256 times. 
    // For number N > 256 it needs to run x iterations for each parameter (i, j, k).
    // where inner_iterations = x. 
    int inner_iterations = range_size / iterations;

    int *results = (int *) malloc(results_size * sizeof(int));
    unsigned int *frequencies = (unsigned int *) malloc(4294967296 * sizeof(int));

    // For timing
    start = clock();

    start_brute_force(results, frequencies, inner_iterations);
    
    // For timing
    end = clock();
    run_time = ((double) (end - start)) / CLOCKS_PER_SEC;

    unsigned long long int temp_count = 0;

    // Prints out the frequency of each results
    for (int i=0; i < range_size; i++) {

        printf("%d: %d \n", i, frequencies[i]);


        temp_count += frequencies[i];
    }

    printf("%llu \n", temp_count);
    printf("Run time: %f \n", run_time);

    free(results);
    free(frequencies);
    return 0;
}
